#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <iostream>
#include <fstream>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>


#define SIZE 9

typedef uint8_t byte;

struct Sudoku
{
	byte board[SIZE][SIZE];
	uint16_t rowNumbers[SIZE];
	uint16_t colNumbers[SIZE];
	uint16_t cellNumbers[SIZE];
	byte rowCounts[SIZE];
	byte colCounts[SIZE];
	byte cellCounts[SIZE];

	//bool active;

	Sudoku(byte board[SIZE][SIZE], uint16_t rowNumbers[SIZE], uint16_t colNumbers[SIZE], 
		uint16_t cellNumbers[SIZE], byte rowCounts[SIZE], byte colCounts[SIZE], byte cellCounts[SIZE])//,bool active)
	{
		//this->active = active;

		for (int i = 0; i < SIZE; i++)
		{
			this->rowNumbers[i] = rowNumbers[i];
			this->colNumbers[i] = colNumbers[i];
			this->cellNumbers[i] = cellNumbers[i];
			this->rowCounts[i] = rowCounts[i];
			this->colCounts[i] = colCounts[i];
			this->cellCounts[i] = cellCounts[i];
			for (int j = 0; j < SIZE; j++)
				this->board[i][j] = board[i][j];
		}
	}
};

__host__ __device__ void PrintSudoku(byte arr[SIZE][SIZE])
{
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			printf("%d ", (int)arr[i][j]);
			if (j == SIZE - 1)
				printf("\n");
		}
	}
	printf("\n");
}

__host__ __device__ byte GetBestCount(byte structure[])
{
	char max = -1;
	char index = -1;

	for (byte i = 0; i < SIZE; i++)
	{
		if (structure[i] > max && structure[i] < 9) 
		{
			max = structure[i];
			index = i;
		}
	}

	return (byte)index;
}

__host__ __device__ byte GetBestCountInRow(byte board[SIZE][SIZE], byte columnCounts[], byte row)
{
	char max = -1;
	char index = -1;

	for (byte j = 0; j < SIZE; j++)
	{
		if (board[row][j] == 0 && columnCounts[j] > max)
		{
			max = (char)columnCounts[j];
			index = j;
		}
	}

	return (byte)index;
}

__host__ __device__ bool IsNumberInRowOrColumn(uint16_t structure, byte number)
{
	return structure & (1U << number);
}

__host__ __device__ void AddNumberToRowOrColumn(uint16_t& structure, byte number)
{
	if (number != 0)
	{
		structure |= (1U << number);
	}
}

__host__ __device__ void RemoveNumberFromRowOrColumn(uint16_t& structure, byte number)
{
	if (number != 0)
	{
		structure &= ~(1U << number);
	}
}

__host__ __device__ void PrintRowsOrColumns(uint16_t structure[])
{
	for (byte i = 0; i < SIZE; i++)
	{
		for (byte number = 1; number <= SIZE; number++)
		{
			printf("%d ", (int)IsNumberInRowOrColumn(structure[i], number));
			if (number % (SIZE + 1) == SIZE)
				printf("\n");
		}
	}
	printf("\n");
}

__host__ __device__ byte cell(byte i, byte j)
{
	return (i / 3) * 3 + j / 3;
}

__global__ void activeResetKernel(bool* d_active, int n)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i >= n)
		return;

	d_active[i] = false;
}

__global__ void copyKernel(Sudoku* d_sudokus, Sudoku* d_sudokus_target, bool* d_active, int* d_active_scan, int n, int newMax, bool lastActive)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i == 0)
		return;

	if ((i < n - 1 && d_active_scan[i] != d_active_scan[i + 1]) || (i == n - 1 && lastActive))
	{
		//printf("copyKernel ------------- %d  - max %d\n", i, n);
		//printf("Swapping %d to %d\n", i, d_active_scan[i]);
		d_sudokus_target[d_active_scan[i]] = d_sudokus[i];

		if (d_active_scan[i] != newMax || (i == n - 1 && lastActive)) {
			//printf("Activating %d\n", d_active_scan[i]);
			d_active[d_active_scan[i]] = true;
		}
	}
}

__global__ void sudokuKernel(Sudoku* d_sudokus, bool* d_active, int n)
{
	//while(1) {
	int i = (blockIdx.x * blockDim.x) +  threadIdx.x;
	if (i > n)
		return;

	if (d_active[0] == true)
	{
		return;
	}

	Sudoku mySudoku = d_sudokus[i];
	if (d_active[i] == false)
	{
		//continue; 
		return;
	}
	//printf("Id: %d\n", i);

	//if (i == 1)
	//	PrintSudoku(mySudoku.board);
	//PrintRowsOrColumns(mySudoku.rowNumbers);
	//return;
	byte row = GetBestCount(mySudoku.rowCounts);
	if (row == (byte)-1) 
	{
		//koniec
		d_sudokus[0] = mySudoku;
		d_active[0] = true;
		return;
	}
	byte col = GetBestCountInRow(mySudoku.board, mySudoku.colCounts, row);
	byte cellnr = cell(row, col);
	for (byte number = 1; number <= SIZE; number++)
	{
		if (!IsNumberInRowOrColumn(mySudoku.rowNumbers[row], number) &&
			!IsNumberInRowOrColumn(mySudoku.colNumbers[col], number) &&
			!IsNumberInRowOrColumn(mySudoku.cellNumbers[cellnr], number))
		{
			mySudoku.board[row][col] = number;

			AddNumberToRowOrColumn(mySudoku.rowNumbers[row], number);
			AddNumberToRowOrColumn(mySudoku.colNumbers[col], number);
			AddNumberToRowOrColumn(mySudoku.cellNumbers[cellnr], number);
			mySudoku.rowCounts[row]++;
			mySudoku.colCounts[col]++;
			mySudoku.cellCounts[cellnr]++;

			//PrintSudoku(d_sudokus[i].board);
			// int size = d_stack[n];
			// int index = d_stack[size - 1];
			// d_stack[n]--;

			int index = n + 1 + (i - 1) * SIZE + (number - 1);
			//printf("Tid:%d, (%d, %d), number %d, activates tid %d\n", i, (int)row, (int)col, (int)number, index);
			d_sudokus[index] = mySudoku;
			d_active[index] = true;
			RemoveNumberFromRowOrColumn(mySudoku.rowNumbers[row], number);
			RemoveNumberFromRowOrColumn(mySudoku.colNumbers[col], number);
			RemoveNumberFromRowOrColumn(mySudoku.cellNumbers[cellnr], number);
			mySudoku.rowCounts[row]--;
			mySudoku.colCounts[col]--;
			mySudoku.cellCounts[cellnr]--;
			mySudoku.board[row][col] = 0;
		}
	}
	d_active[i] = false;
	//}
}

void ReadSudoku(byte arr[SIZE][SIZE], std::string filename)
{
	std::ifstream stream(filename);

	char c = stream.get();
	byte i = 0, j = 0;

	while (stream.good() && c != '\n') 
	{
		if (c != 'x')
		{
			int n = atoi(&c);
			arr[i][j] = n;
		}
		else
			arr[i][j] = 0;
		c = stream.get();
		j++;
		if (j == SIZE)
		{
			j = 0;
			i++;
		}
	}
	stream.close();
}

void GetRowColNumbers(byte sudoku[SIZE][SIZE], uint16_t rows[], uint16_t columns[], uint16_t areas[])
{
	for (byte i = 0; i < SIZE; i++)
	{
		rows[i] = 0;
		columns[i] = 0;
		areas[i] = 0;
	}

	for (byte i = 0; i < SIZE; i++)
	{
		for (byte j = 0; j < SIZE; j++)
		{
			if (sudoku[i][j] != 0)
			{
				AddNumberToRowOrColumn(rows[i], sudoku[i][j]);
				AddNumberToRowOrColumn(columns[j], sudoku[i][j]);
				AddNumberToRowOrColumn(areas[cell(i, j)], sudoku[i][j]);
			}
		}
	}
}

void GetRowColCounts(uint16_t rows[], uint16_t columns[], uint16_t cells[], byte rowCounts[], byte columnCounts[], byte cellCounts[])
{
	for (byte i = 0; i < SIZE; i++)
	{
		rowCounts[i] = 0;
		columnCounts[i] = 0;
		cellCounts[i] = 0;
	}

	for (byte i = 0; i < SIZE; i++)
	{
		for (byte number = 1; number <= SIZE; number++)
		{
			if (IsNumberInRowOrColumn(rows[i], number))
				rowCounts[i]++;
			if (IsNumberInRowOrColumn(columns[i], number))
				columnCounts[i]++;
			if (IsNumberInRowOrColumn(cells[i], number))
				cellCounts[i]++;
		}
	}
}

hipError_t PrepareSudoku(byte sudokuArray[SIZE][SIZE])
{
	//PrintSudoku(sudokuArray);

	uint16_t rowNumbers[SIZE];
	uint16_t colNumbers[SIZE];
	uint16_t cellNumbers[SIZE];	
	byte rowCounts[SIZE];
	byte colCounts[SIZE];
	byte cellCounts[SIZE];

	GetRowColNumbers(sudokuArray, rowNumbers, colNumbers, cellNumbers);
	GetRowColCounts(rowNumbers, colNumbers, cellNumbers, rowCounts, colCounts, cellCounts);

	//PrintRowsOrColumns(rowNumbers);
	//PrintRowsOrColumns(colNumbers);
	//PrintRowsOrColumns(cellNumbers);

	hipError_t cudaStatus;

	Sudoku activeSudoku(sudokuArray, rowNumbers, colNumbers, cellNumbers, rowCounts, colCounts, cellCounts);

	int nBlocks = 1000000;
	int activeBlocks = 1;
	Sudoku *d_sudokus;
	Sudoku *d_sudokus_target;

	Sudoku *h_sudokus = (Sudoku*)malloc((activeBlocks + 1) * sizeof(Sudoku));
	if (h_sudokus == NULL) {
		fprintf(stderr, "malloc failed!");
		return cudaStatus;
	}
	bool *d_active;
	int *d_active_scan;

	bool *h_active = (bool*)malloc((activeBlocks + 9 * activeBlocks + 1) * sizeof(bool));
	if (h_active == NULL) {
		fprintf(stderr, "malloc failed!");
		return cudaStatus;
	}

	for (int i = 0; i < (activeBlocks + 1); i++) {
		if (i == 1)
		{
			h_sudokus[i] = activeSudoku;
		}
	}

	for (int i = 0; i < (activeBlocks + 9 * activeBlocks + 1); i++) {
		if (i == 1)
		{
			h_active[i] = true;
		}
		else
		{
			h_active[i] = false;
		}
	}

	cudaStatus = hipMalloc((void**)&d_sudokus, (activeBlocks + 9 * activeBlocks + 1) * sizeof(Sudoku));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(d_sudokus);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(d_sudokus, h_sudokus, (activeBlocks + 1) * sizeof(Sudoku), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(d_sudokus);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&d_active, (activeBlocks + 9 * activeBlocks + 1) * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(d_sudokus);
		hipFree(d_active);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&d_active_scan, (activeBlocks + 9 * activeBlocks + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(d_sudokus);
		hipFree(d_active);
		hipFree(d_active_scan);
		return cudaStatus;
	}


	cudaStatus = hipMemcpy(d_active, h_active, (activeBlocks + 9 * activeBlocks + 1) * sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(d_sudokus);
		hipFree(d_active);
		hipFree(d_active_scan);
		return cudaStatus;
	}
	// Launch a kernel on the GPU with one thread for each element.

	thrust::device_ptr<bool> dev_active_ptr(d_active);
	thrust::device_ptr<int> dev_active_scan_ptr(d_active_scan);

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	int i = 0;
	bool lastActive;
	while (1)
	//for (int i = 0; i < 3; i++)
	{
		i++;
		//printf("Iteration: %d\n", i); // 1 3
		sudokuKernel <<<(activeBlocks + 1)/1024 + 1, 1024>>>(d_sudokus, d_active, activeBlocks);
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				hipFree(d_sudokus);
				hipFree(d_active);
				hipFree(d_active_scan);
				return cudaStatus;
			}

			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching sudokuKernel!\n", cudaStatus);
				hipFree(d_sudokus);
				hipFree(d_active);
				hipFree(d_active_scan);
				return cudaStatus;
			}

		if (dev_active_ptr[0] == true)
		{
			printf("Solution found!\n");
			break;
		}

		activeBlocks = activeBlocks + activeBlocks * 9;


		//printf("Scanning table, length: %d\n", activeBlocks + 1);
		dev_active_ptr[0] = true;
		thrust::exclusive_scan(dev_active_ptr, dev_active_ptr + activeBlocks + 1, dev_active_scan_ptr);
		dev_active_ptr[0] = false;
		int newActive = thrust::max_element(dev_active_scan_ptr, dev_active_scan_ptr + activeBlocks + 1)[0];

		lastActive = dev_active_ptr[activeBlocks];
		if (lastActive)
			newActive++;

		//printf("New active: %d\n", newActive);

		//printf("Allocing table, length: %d\n", activeBlocks + 9 * activeBlocks + 1);
		cudaStatus = hipMalloc((void**)&d_sudokus_target, (activeBlocks + 9 * activeBlocks + 1) * sizeof(Sudoku));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree(d_sudokus);
			return cudaStatus;
		}

		hipFree(d_active);
		cudaStatus = hipMalloc((void**)&d_active, (activeBlocks + 9 * activeBlocks + 1) * sizeof(bool));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree(d_sudokus);
			hipFree(d_active);
			return cudaStatus;
		}
		dev_active_ptr = thrust::device_ptr<bool>(d_active);

		activeResetKernel <<<(activeBlocks + 9 * activeBlocks + 1) / 1024 + 1, 1024 >>>(d_active, (activeBlocks + 9 * activeBlocks + 1));
		// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "activeResetKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				hipFree(d_sudokus);
				hipFree(d_active);
				hipFree(d_active_scan);
				return cudaStatus;
			}

			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching activeResetKernel!\n", cudaStatus);
				hipFree(d_sudokus);
				hipFree(d_active);
				hipFree(d_active_scan);
				return cudaStatus;
			}

		copyKernel <<<(activeBlocks + 1) / 1024 + 1, 1024 >>>(d_sudokus, d_sudokus_target, d_active, d_active_scan, (activeBlocks + 1), newActive, lastActive);
		// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "copyKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				hipFree(d_sudokus);
				hipFree(d_active);
				hipFree(d_active_scan);
				return cudaStatus;
			}

			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching copyKernel!\n", cudaStatus);
				hipFree(d_sudokus);
				hipFree(d_active);
				hipFree(d_active_scan);
				return cudaStatus;
			}

		hipFree(d_active_scan);
		cudaStatus = hipMalloc((void**)&d_active_scan, (activeBlocks + 9 * activeBlocks + 1) * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree(d_sudokus);
			hipFree(d_active);
			hipFree(d_active_scan);
			return cudaStatus;
		}
		dev_active_scan_ptr = thrust::device_ptr<int>(d_active_scan);

		hipFree(d_sudokus);
		d_sudokus = d_sudokus_target;

		activeBlocks = newActive - 1;
		//getchar();
	}

	cudaStatus = hipMemcpy(h_sudokus, d_sudokus, 1 * sizeof(Sudoku), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(d_sudokus);
		return cudaStatus;
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);

	printf("Original:\n");
	PrintSudoku(sudokuArray);
	printf("Solved:\n");
	PrintSudoku(h_sudokus[0].board);

	hipFree(d_sudokus);
	hipFree(d_active);
	hipFree(d_active_scan);
	return cudaStatus;
}

int main()
{
	byte sudoku[SIZE][SIZE];

	printf("Entry:\n");
	ReadSudoku(sudoku, "Entry.txt");
	hipError_t cudaStatus = PrepareSudoku(sudoku);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "PrepareSudoku failed!");
		return 1;
	}
	printf("------------------------------------------------------------\n");

	printf("Easy:\n");
	ReadSudoku(sudoku, "Easy.txt");
	cudaStatus = PrepareSudoku(sudoku);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "PrepareSudoku failed!");
		return 1;
	}
	printf("------------------------------------------------------------\n");

	printf("Medium:\n");
	ReadSudoku(sudoku, "Medium.txt");
	cudaStatus = PrepareSudoku(sudoku);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "PrepareSudoku failed!");
		return 1;
	}
	printf("------------------------------------------------------------\n");

	printf("Hard:\n");
	ReadSudoku(sudoku, "Hard.txt");
	cudaStatus = PrepareSudoku(sudoku);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "PrepareSudoku failed!");
		return 1;
	}
	printf("------------------------------------------------------------\n");

	printf("Evil:\n");
	ReadSudoku(sudoku, "Evil.txt");
	cudaStatus = PrepareSudoku(sudoku);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "PrepareSudoku failed!");
		return 1;
	}
    return 0;
}